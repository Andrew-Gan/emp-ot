#include "emp-ot/ferret/aes_expand.cuh"
#include "emp-tool/emp-tool.h"

#define AES_KEYSIZE 176

void gpu_ggm_tree_send(GPUvector<OTblock>& leftSum, GPUvector<OTblock>& rightSum,
    GPUvector<OTblock>& ggm_tree, GPUdata& secret_sum, const GPUdata& secret, int depth) {
    
    uint32_t k0_blk[4] = {3242342};
    uint32_t k1_blk[4] = {8993849};
    AesHash aesHash((uint8_t*) k0_blk, (uint8_t*) k1_blk);
    GPUvector<OTblock> separated(2 * numOT);

    for (uint64_t d = 1, w = 2; d <= depth; d++, w *= 2) {
        aesHash.expand(ggm_tree.data(w-1), separated, gmm_tree.data(w/2-1), w);
        separated.sum_async(2, w / 2);
        hipMemcpy(leftSum.data(d-1), separated.data(0));
        hipMemcpy(rightSum.data(d-1), separated.data(1));
    }

    // memset(secretSum, 0, sizeof(secretSum));
    // OTBlock one = { .data = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFE} };
    // OTBlock *one_d;
    // hipMalloc(&one_d, sizeof(*one_d));
    // hipMemcpy(one_d, &one, sizeof(*one_d), hipMemcpyHostToDevice);

    // ggm_tree.and_scalar(one_d);
    // GPUVector<OTBlock> nodes_sum(leave_n + 1);
    // nodes_sum = ggm_tree;
    // nodes_sum.set(leave_n, secret);
    // nodes_sum.sum(1, leave_n+1);
    // secret_sum = nodes_sum.data(0);
}

void gpu_ggm_tree_recv(GPUvector<OTblock>& ggm_tree, bool *choices,
    const GPUvector<OTblock>& sums, const GPUdata& secret_sum, uint64_t choice_pos) {
    
    uint32_t k0_blk[4] = {3242342};
    uint32_t k1_blk[4] = {8993849};
    AesHash aesHash((uint8_t*) k0_blk, (uint8_t*) k1_blk);
    GPUvector<OTblock> separated(2 * numOT);
    uint64_t activeParent = 0;
    uint8_t choice;
    uint64_t offset;

    for (uint64_t d = 1, w = 2; d <= depth; d++, w *= 2) {
        aesHash.expand(ggm_tree, separated, *inBuffer, w);
        choice = choices[d-1];
        offset = (w / 2) * choice + activeParent;
        hipMemcpy(separated.data(offset), sums.data(d-1), sizeof(OTblock), hipMemcpyDeviceToDevice);
        if (d == depth) {
            offset = (w / 2) * (1-choice) + activeParent;
            hipMemcpy(separated.data(offset), sums.data(d), sizeof(OTblock), hipMemcpyDeviceToDevice);
        }
        separated.sum_async(2, w / 2);
        offset = 2 * activeParent + choice;
        hipMemcpy(outBuffer->data(offset), separated.data(choice), sizeof(OTblock), hipMemcpyDeviceToDevice);
        if (d == depth) {
            offset = 2 * activeParent + (1-choice);
            hipMemcpy(outBuffer->data(offset), separated.data(1-choice), sizeof(OTblock), hipMemcpyDeviceToDevice);
        }

        activeParent *= 2;
        activeParent += 1 - choice;
    }

    // hipMemset(ggm_tree.data(choice_pos), 0, sizeof(OTBlock));
    // OTBlock one = { .data = {0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFF, 0xFFFFFFFE} };
    // OTBlock *one_d;
    // hipMalloc(&one_d, sizeof(*one_d));
    // hipMemcpy(one_d, &one, sizeof(*one_d), hipMemcpyHostToDevice);

    // ggm_tree.and_scalar(one_d);
    // GPUVector<OTBlock> nodes_sum(leave_n + 1);
    // nodes_sum = ggm_tree;
    // nodes_sum.set(leave_n, secret_sum);
    // nodes_sum.sum(1, leave_n+1);
    // ggm_tree.set(choice_pos, nodes_sum.data(0));
}